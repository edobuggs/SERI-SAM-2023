#include "hip/hip_runtime.h"
/*
The codec is compiled into .dll format such that it can be integrated into
LabView.
This code aims to perform synthetic aperture beamforming for annular array.
The flow of this code is as follow:
1. Copy data from host memory to GPU memory
2. Perform analytic signal conversion
3. Perform delay and sum (using geometric approach instead of phase)
4. Copy image from GPU to host memory

Data input:
char * RFdata			: RF data acquired from ADC
int numScanline			: Number of scanline
int dataVectorLength	: Number of sample per channel
float * tdrGeometry		: Annular array transducer geometry
int numChannel			: Number of element of the array
int fs					: Sampling frequency

Image Output:
short * RFimage					: Beamformed image

Packing of data/image:	see documentation P.?
*/
extern "C" __declspec (dllexport) int SAbeamformer(short * hostRFimage, char * hostRFdata, int numScanline, int dataVectorLength, float * hostTdrGeometry, int numChannel, int * hostTxPos, int *hostRxPos, int numTxRxPair, float * hostWeighting, int fs, int sos, float * time_start);

// define constants
#define FILTERLENGTH 51
#define ASCBLOCKSIZE 256
#define BFBLOCKSIZE 512

//#define DEBUG_MODE




// includes
#include <stdio.h> // for reading files
#include <conio.h>
#include <Windows.h>
#include <stdlib.h> // for memory allocation
#include <math.h>
//#include "riversideSA.h"
//inlcudes

#include <hip/hip_runtime_api.h>


/* beamforming
input:
------------------------------------------------------------
short * RFdata_I		: real part of the RF data
short * RFdata_Q		: imag part of the RF data
int dataVectorLength: number of sample per received channel
float * channelDelay	: transducer geometry
int numChannel		: number of channel of the transducer
int fs				: sampling frequency
------------------------------------------------------------

output:
------------------------------------------------------------
short * RFimage		: beamformed image
------------------------------------------------------------

*/
__global__ void beamforming (short * RFimage, short * RFdata_I, short * RFdata_Q, int dataVectorLength, float * channelDelay, int numChannel, int *txPos, int *rxPos, int numTxRxpair, float * weighting, int fs, int imageLength, int sos, float * time_start)
{
	volatile int idx = blockIdx.y * blockDim.x + threadIdx.x;
	int currentScanline = blockIdx.x;
	int numScanline = gridDim.x;
	volatile float path = 0;
	volatile int path_floor = 0;
	volatile float sum = 0;
	volatile float sum_hil = 0;
	volatile float ratio = 0;
	if (idx<imageLength) {
		// the idx is within the image array limit
		for (int currentPair=0;currentPair<numTxRxpair;currentPair++)
		{
			// Compute the total tx/rx path for that pixel
			path = ( channelDelay[txPos[currentPair]*imageLength + idx] + channelDelay[rxPos[currentPair]*imageLength + idx])*fs/sos - time_start[currentPair] * fs;
			if ( (path < dataVectorLength - 1)  && (path > 0) )
			{
				// the total path is within data range
				path = path + currentScanline*dataVectorLength + currentPair*dataVectorLength*numScanline;
				path_floor = floorf(path);
				ratio = path - path_floor;

				sum = sum + weighting[currentPair] * ( (1-ratio)*RFdata_I[path_floor] + ratio*RFdata_I[path_floor + 1] );
				sum_hil = sum_hil + weighting[currentPair] * ( (1-ratio)*RFdata_Q[path_floor] + ratio*RFdata_Q[path_floor + 1] );
			}
		}
		RFimage[idx + currentScanline*imageLength] = 20.f*log10f(sqrt(sum*sum + sum_hil*sum_hil));
	}
}

/* calChannelDelay
input:
------------------------------------------------------------
int imageLength		: number of pixel per scan-line
int numChannel		: number of channel of the transducer
float * tdrGeometry	: transducer geometry
int fs				: sampling frequency
------------------------------------------------------------

output:
------------------------------------------------------------
float * channelDelay: array holding delay of each channel
------------------------------------------------------------
*/
__global__ void calChannelDelay(float * channelDelay, int imageLength, int numChannel, float * tdrGeometry, int fs, int sos, float time_start)
{
	volatile int idx = blockIdx.y * blockDim.x + threadIdx.x;
	float depth;
	float extra_depth = tdrGeometry[numChannel*2];
	if (idx<imageLength) {
		// the idx is within the image array limit
		depth = (idx*1.f*sos/fs + time_start*sos)/2;
		for (int currentChannel=0;currentChannel<numChannel;currentChannel++)
		{
			channelDelay[imageLength*currentChannel + idx] = sqrtf( tdrGeometry[currentChannel]*tdrGeometry[currentChannel] + (depth + extra_depth + tdrGeometry[numChannel + currentChannel])*(depth + extra_depth + tdrGeometry[numChannel + currentChannel]) );
		}
	}
}

/* demodulation
input:
------------------------------------------------------------
short * RFdata		: input RF data
int dataVectorLength: number of sample per received channel
int numScanline		: number of scanline acquired
int numChannel		: number of channel of the transducer
float * filter		: filter coefficient for analytic signal conversion
------------------------------------------------------------

output:
------------------------------------------------------------
short * RFdata_I		: beamformed image
short * RFdata_Q		: beamformed image
------------------------------------------------------------

*/

__global__ void demodulation (short * RFdata_I, short * RFdata_Q, char * RFdata, int dataVectorLength, int numScanline, int numChannel, float * filter, float * meanVector)
{
	__shared__ float coeff[FILTERLENGTH];
	if (threadIdx.x < FILTERLENGTH) {
		// the thread ID is between 0 and filter order - 1
		coeff[threadIdx.x] = filter[threadIdx.x];
	}
	__syncthreads();

	volatile int z = 0;
	volatile int i = 0;
	volatile float sum_hil = 0;
	volatile int j = 0;
	volatile int pos = (blockIdx.y*gridDim.x+blockIdx.x)*dataVectorLength;

	for (i=0;i<(dataVectorLength/blockDim.x+1);i++) {
		z = i*blockDim.x + threadIdx.x;
		sum_hil=0;
		if (z<(dataVectorLength-FILTERLENGTH-1)) {
			for (j=0;j<FILTERLENGTH;j++) {
				sum_hil += coeff[j]*(float) RFdata[z + pos + j];
			}
			RFdata_I [z + pos + FILTERLENGTH/2] = (short) RFdata [z + pos + FILTERLENGTH/2] - meanVector[blockIdx.x + gridDim.x * blockIdx.y];
			RFdata_Q [z + pos + FILTERLENGTH/2] = -sum_hil - meanVector[blockIdx.x + gridDim.x * blockIdx.y];
			//printf("sum_hil = %.4f\n",sum_hil);
		}
	}
}


__global__ void meanFilter(char * RFdata, int dataVectorLength, int numScanline, int numChannel, float *meanVector)
{
	__shared__ volatile float mean[ASCBLOCKSIZE];
	float tmp_mean = 0;
	volatile int i, z;
	volatile int pos = (blockIdx.y*gridDim.x+blockIdx.x)*dataVectorLength;

	// Divide the data vector into segments, each thread will sum up the same data point from different segments
	for (i=0;i<(dataVectorLength/blockDim.x+1);i++) 
	{
		z = i*blockDim.x + threadIdx.x;
		if (z<(dataVectorLength)) 
		{
			tmp_mean += RFdata[z + pos];
		}
	}
	// store the temporary result into shared memory
	mean[threadIdx.x] = tmp_mean;
	__syncthreads();
	// Sum up the temporary result using Reduction method
	if (threadIdx.x < 128)
		mean[threadIdx.x] += mean[threadIdx.x + 128];
	__syncthreads();
	if (threadIdx.x < 64)
		mean[threadIdx.x] += mean[threadIdx.x + 64];
	__syncthreads();
	if (threadIdx.x < 32)
		mean[threadIdx.x] += mean[threadIdx.x + 32];
	__syncthreads();
	if (threadIdx.x < 16)
		mean[threadIdx.x] += mean[threadIdx.x + 16];
	if (threadIdx.x < 8)
		mean[threadIdx.x] += mean[threadIdx.x + 8];
	if (threadIdx.x < 4)
		mean[threadIdx.x] += mean[threadIdx.x + 4];
	if (threadIdx.x < 2)
		mean[threadIdx.x] += mean[threadIdx.x + 2];
	if (threadIdx.x < 1)
	{
		mean[threadIdx.x] += mean[threadIdx.x + 1];
		meanVector[blockIdx.x + gridDim.x * blockIdx.y] = mean[threadIdx.x]/dataVectorLength;
	}
}

int SAbeamformer(short * hostRFimage, char * hostRFdata, int numScanline, int dataVectorLength, float * hostTdrGeometry, int numChannel, int * hostTxPos, int * hostRxPos, int numTxRxpair, float * hostWeighting, int fs, int sos, float * hostTimeStart)
{
	// Select GPU - Default: choose cuda device 0 (you can use deviceQuery to check which card is device 0)
	hipSetDevice(0);

	// Step 1. ALLOCATE ENOUGH MEMORY (both Host and GPU)
	// 1a. Host:
	// Filter
	
	//Create Console window for debug
#ifdef DEBUG_MODE
	AllocConsole();
	
#endif

	float * hostFilter;
	int filterSize = FILTERLENGTH * sizeof(float);
	hostFilter = (float *) malloc(filterSize);
	
	FILE *fp;
	fp = fopen("hilbert.bin","rb+");
	if (!fp)
	{
		// cannot find or open hilbert.bin
#ifdef DEBUG_MODE
		_cprintf("cannot load hilbert.bin\n");
		while(!kbhit());
		FreeConsole();
#endif
		return -2;
	}
	fread(hostFilter, 1, filterSize, fp);
	fclose(fp);
#ifdef DEBUG_MODE
	_cprintf("filterSize = %d\n", filterSize);
	_cprintf("hostFilter...%.4f\n",hostFilter[26]);
#endif

	// 1b. GPU:

	char * RFdata;
	short *RFdata_I, * RFdata_Q, *RFimage;
	float * filter, * tdrGeometry, * channelDelay, * weighting, *time_start, * meanVector;
	int * txPos, * rxPos;
	int dataSize_char = dataVectorLength*numScanline*numChannel*numChannel*sizeof(char);
	int dataSize_short = dataVectorLength*numScanline*numChannel*numChannel*sizeof(short);
	int imageLength = dataVectorLength;
	//_cprintf("image length = %d\n", imageLength);
	int imageSize = imageLength*numScanline*sizeof(short);
	int tdrGeometrySize = (2*numChannel+1)*sizeof(float);
	int channelDelaySize = numTxRxpair * imageLength * sizeof(float);
	int posSize = numTxRxpair * sizeof(int);

	
	// RFdata
	hipMalloc( (void **) &RFdata, dataSize_char);
	// RFdata_I
	hipMalloc( (void **) &RFdata_I, dataSize_short);
	// RFdata_Q
	hipMalloc( (void **) &RFdata_Q, dataSize_short);
	// RFimage
	hipMalloc( (void **) &RFimage, imageSize);
	// filter
	hipMalloc( (void **) &filter, filterSize);
	// tdrGeometry
	hipMalloc( (void **) &tdrGeometry, tdrGeometrySize );
	// channelDelay
	hipMalloc( (void **) &channelDelay, channelDelaySize);
	// weighting
	hipMalloc( (void **) &weighting, posSize); // weighting must match with the number of TxRxpair
	// txPos
	hipMalloc( (void **) &txPos, posSize);
	// rxPos
	hipMalloc( (void **) &rxPos, posSize);
	// time start
	hipMalloc( (void **) &time_start, posSize);
	// meanVector
	hipMalloc ( (void **) & meanVector, sizeof(float) * numScanline * numTxRxpair );

	hipMemset( RFdata_I, 0, dataSize_short);
	hipMemset( RFdata_Q, 0, dataSize_short);
	hipMemset( RFimage, 0, imageSize);
	hipMemset(tdrGeometry, 0, tdrGeometrySize);
	hipMemset(weighting, 0, posSize);
	hipMemset(txPos, 0, posSize);
	hipMemset(rxPos, 0, posSize);
	hipMemset(time_start, 0, posSize);
	hipMemset( meanVector, 0, numScanline * numTxRxpair * sizeof(float) );

#ifdef DEBUG_MODE
	_cprintf("Step 1 done\n");
#endif

	// Step 2. COPY INPUT DATA TO GPU
	// 2a. load ASC filter from file
	hipMemcpy(filter, hostFilter, filterSize, hipMemcpyHostToDevice);
	// 2b. Copy input data to GPU
	hipMemcpy(RFdata, hostRFdata, dataSize_char, hipMemcpyHostToDevice);
	// 2c. Copy tdrGeometry
	hipMemcpy(tdrGeometry, hostTdrGeometry, tdrGeometrySize, hipMemcpyHostToDevice);
	// 2d. Copy txPos
	hipMemcpy(txPos, hostTxPos, posSize, hipMemcpyHostToDevice);
	// 2e. Copy rxPos
	hipMemcpy(rxPos, hostRxPos, posSize, hipMemcpyHostToDevice);
	// 2f. Copy weighting
	hipMemcpy(weighting, hostWeighting, posSize, hipMemcpyHostToDevice);
	// 2g. Copy time_start
	hipMemcpy(time_start, hostTimeStart, posSize, hipMemcpyHostToDevice);

#ifdef DEBUG_MODE
	_cprintf("Step 2 done\n");
#endif
	// Step 3a. PERFORM MEAN SUBTRACTION FILTERING
	dim3 dimGridASC(numScanline, numTxRxpair);
	meanFilter <<< dimGridASC, ASCBLOCKSIZE >>> (RFdata, dataVectorLength, numScanline, numChannel, meanVector);

#ifdef DEBUG_MODE
	FILE *fp_mean;
	fp_mean = fopen("meanVector.bin","wb+");
	float * hostMeanVector;
	hostMeanVector = (float *) malloc ( numScanline * numTxRxpair * sizeof(float) );
	hipMemcpy(hostMeanVector, meanVector, numScanline * numTxRxpair * sizeof(float), hipMemcpyDeviceToHost );
	fwrite(hostMeanVector, 1, sizeof(float) * numScanline * numTxRxpair, fp_mean);
	fclose(fp_mean);
#endif

	// Step 3b. PERFORM ANALYTIC SIGNAL CONVERSION
	demodulation <<< dimGridASC, ASCBLOCKSIZE >>> (RFdata_I, RFdata_Q, RFdata, dataVectorLength, numScanline, numChannel, filter, meanVector);

#ifdef DEBUG_MODE
	_cprintf("Step 3 done\n");
#endif
	
	/*
	// save ASC output for debugging
	short *hostRFdata_I;
	hostRFdata_I = (short *) malloc (dataSize_short);
	hipMemcpy(hostRFdata_I, RFdata_Q, dataSize_short, hipMemcpyDeviceToHost);
	fp = fopen("RFdata_Q.bin","wb+");
	fwrite(hostRFdata_I, 1, dataSize_short, fp);
	fclose(fp);

	free(hostRFdata_I);
	*/
	
	// Step 4a. CALCULATE DELAYS FOR EACH CHANNEL
	int tmpSize =  ceilf( (float) imageLength/BFBLOCKSIZE );
	dim3 dimGridCalDelay(1, tmpSize );
	calChannelDelay <<< dimGridCalDelay, BFBLOCKSIZE >>> (channelDelay, imageLength, numChannel, tdrGeometry, fs, sos, hostTimeStart[0]);

#ifdef DEBUG_MODE
	_cprintf("Step 4a done\n");
#endif

	/*
	// save channelDelay for debugging
	float *hostDelay;
	hostDelay = (float *) malloc (channelDelaySize);
	hipMemcpy(hostDelay, channelDelay, channelDelaySize, hipMemcpyDeviceToHost);
	fp = fopen("channelDelay.bin","wb+");
	fwrite(hostDelay, 1, channelDelaySize, fp);
	fclose(fp);

	free(hostDelay);
	*/

	// Step 4b. PERFORM SA BEAMFORMING
	dim3 dimGridBF(numScanline, tmpSize );
	beamforming <<< dimGridBF, BFBLOCKSIZE >>> (RFimage, RFdata_I, RFdata_Q, dataVectorLength, channelDelay, numChannel, txPos, rxPos, numTxRxpair, weighting, fs, imageLength, sos, time_start);

#ifdef DEBUG_MODE
	_cprintf("Step 4b done\n");
#endif

	// Step 5. COPY IMAGE TO HOST
	hipMemcpy(hostRFimage, RFimage, imageSize, hipMemcpyDeviceToHost);

	// save RF image for debugging
#ifdef DEBUG_MODE
	fp = fopen("RFimage.bin","wb+");
	fwrite(hostRFimage, 1, imageSize, fp);
	fclose(fp);
#endif

	// Step 6. FREE UP MEMORY
	// 6a. Host memory
	free(hostFilter);

	// 6b. GPU memory
	hipFree(RFdata);
	hipFree(RFdata_I);
	hipFree(RFdata_Q);
	hipFree(RFimage);
	hipFree(filter);
	hipFree(tdrGeometry);
	hipFree(channelDelay);
	hipFree(txPos);
	hipFree(rxPos);
	hipFree(weighting);
	hipFree(time_start);
	hipFree(meanVector);

#ifdef DEBUG_MODE
	Sleep(3000);
	FreeConsole();
#endif

	return 0;
}
