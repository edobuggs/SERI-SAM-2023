#include "hip/hip_runtime.h"
/*
This is modified version of codec to apply a digitital filter to the RF data prior to beamforming
J Ketterling adapted it in 2016 or somewhere in that vacinity. No idea.
Just adapted the Hilbert demodulation code to do digital filter. Input for FIR parameters comes in with dll call.
Also added ability to downsample final output, weight each RF line, 

The codec is compiled into .dll format such that it can be integrated into
LabView.
This code aims to perform synthetic aperture beamforming for annular array.
The flow of this code is as follow:
1. Copy data from host memory to GPU memory
2. Perform analytic signal conversion
3. Perform delay and sum (using geometric approach instead of phase)
4. Copy image from GPU to host memory

Data input:
short * RFdata			: RF data acquired from ADC. Use 16 bit to cover any type of digitizer
int numScanline			: Number of scanline
int dataVectorLength	: Number of sample per channel
float * tdrGeometry		: Annular array transducer geometry
int numChannel			: Number of element of the array
int *hostTxPos			: table of transmit element as it relates to order of rows in RFdata 0,0,0,0,0,1,1,1,1, et
int *hostRxPos			: table of receive element as it relates to order of rows in RFdata 0,1,2,3,4,0,1 etc
int numTxRxPair			: total number of Tx-Rcv pairs
float hostWeighting		: weighting of each Tx-Rcv pair
int fs					: Sampling frequency
int sos					: speed of sound
loat * time_start		: delay to start of RF line
float * hostBPfilter	: coefficients of BP filter
int DataFlag			: how to index all of data for beamforming. 0 == five pass with one pulse/pass 1==one pass with 5 pulses/line
float * hostfilter		: coefficients of hilbert filter
int BPFILTERLENGTH		: bandpass filter length
int FILTERLENGTH		: hilbert coefficient length
int downfactor			: downsample output image and data
int ADbits				: bits of AD sample

Image Output:
short * RFimage			: Beamformed image log compressed
short * RF_Out			: Raw beamformed data




Packing of data/image:	see documentation P.?
*/
extern "C" __declspec (dllexport) int SAbeamformerFilter(short * hostRF_Out, short * hostRFimage, short * hostRFdata, int numScanline, int dataVectorLength, float * hostTdrGeometry, int numChannel, int * hostTxPos, int *hostRxPos, int numTxRxPair, float * hostWeighting, int fs, int sos, float * time_start, float * hostBPfilter, int DataFlag, float * hostfilter, int BPFILTERLENGTH, int FILTERLENGTH, int downfactor ,int ADbits);

// define constants
//#define FILTERLENGTH  51
//#define BPFILTERLENGTH  51
#define ASCBLOCKSIZE 256 //this is max number of threads per block for filtering and demodulation. Procsessing done in chunks of filterlength. 
#define BFBLOCKSIZE 512 //max number of threads for delay and beamforming. card dependent. Originally 512

//#define DEBUG_MODE



// includes
#include <stdio.h> // for reading files
#include <conio.h>
#include <Windows.h>
#include <stdlib.h> // for memory allocation
#include <math.h>
//#include "riversideSA.h"
//inlcudes




//#include <hip/hip_runtime_api.h>


/* beamforming
input:
------------------------------------------------------------
short * RFdata_I		: real part of the RF data
short * RFdata_Q		: imag part of the RF data
int dataVectorLength	: number of sample per received channel
float * channelDelay	: transducer geometry
int numChannel			: number of channel of the transducer
int *hostTxPos			: table of transmit element as it relates to order of rows in RFdata 0,0,0,0,0,1,1,1,1, et
int *hostRxPos			: table of receive element as it relates to order of rows in RFdata 0,1,2,3,4,0,1 etc
int numTxRxpair			: number of total beamformed pairs
float * weighting		: weighting of each Tx-Rcv pair
int fs					: sampling frequency
int imageLength			: number of pixel per scan-line
int sos					: speed of sound
loat * time_start		: delay to start of RF line
int DataFlag			: how to index all of data for beamforming. 0 == five pass with one pulse/pass 1==one pass with 5 pulses/line
float maxdB				: maximum dB value for full clipping data range
int downfactor			: downsample output image and data
------------------------------------------------------------

output:
------------------------------------------------------------

Image Output:
short * RFimage			: Beamformed image log compressed
short * RF_Out			: Raw beamformed data
------------------------------------------------------------
*/
__global__ void beamforming(short * RF_Out, short * RFimage, short * RFdata_I, short * RFdata_Q, int dataVectorLength, float * channelDelay, int numChannel, int *txPos, int *rxPos, int numTxRxpair, float * weighting, int fs, int imageLength, int sos, float * time_start, int DataFlag, float maxdB, int downfactor)
{
	volatile int idx = blockIdx.y * blockDim.x + threadIdx.x; //runs across RF pts from one vector block IDx.y is RFps/512


	int currentScanline = blockIdx.x; // runs across rf flines
	int numScanline = gridDim.x; // total # rf lines
	int currentPair = 0;
	volatile float path = 0;
	volatile int path_floor = 0;
	volatile float sum = 0;
	volatile float sum_hil = 0;
	volatile float ratio = 0;


	if (idx < imageLength) { //image length is length of one vector
		// the idx is within the image array limit
		//for (int currentPair = 0; currentPair < numTxRxpair; currentPair++){
		for (int currentRCV = 0; currentRCV < numChannel; currentRCV++){
			for (int currentTX = 0; currentTX < numChannel; currentTX++){
			
			// Compute the total tx/rx path for that pixel
			currentPair = currentRCV * numChannel + currentTX;
			path = (channelDelay[txPos[currentPair] * imageLength + idx] + channelDelay[rxPos[currentPair] * imageLength + idx])*fs / sos - time_start[currentPair] * fs; //find pt shift

				if ((path < dataVectorLength - 1) && (path > 0)){  //can't have negative shift

				// the total path is within data range

				// handle 5 pass or single pass data order. These adjust the indexes used to generate final value of path
					if (DataFlag != 1) { // 5 pass case is default
						path = path + currentScanline*dataVectorLength + currentPair*dataVectorLength*numScanline; //index to pull +start index of each RF line + start index of row. one tx/rcv move
						}
						else { // single pass case. Data in rcv channel blocks interleaved
						path = path + currentScanline*dataVectorLength*numChannel + currentRCV*dataVectorLength + currentTX*dataVectorLength*numScanline*numChannel; //index 
						}

				path_floor = floorf(path);
				ratio = path - path_floor; //find fractional difference in shift

				sum = sum + weighting[currentPair] * ((1 - ratio)*RFdata_I[path_floor] + ratio*RFdata_I[path_floor + 1]); //interpolate between samples. This RF part
				sum_hil = sum_hil + weighting[currentPair] * ((1 - ratio)*RFdata_Q[path_floor] + ratio*RFdata_Q[path_floor + 1]); //interpolate between
				}
			}
		}
			//RFimage[idx + currentScanline*imageLength] = 10*(20.f*log10f(sqrt(sum*sum + sum_hil*sum_hil)+1) -maxdB); //expand scale by 10 becase of data gets forced to integer
			RFimage[idx + currentScanline*imageLength] = 20.f*log10f(sqrt(sum*sum + sum_hil*sum_hil) + 1)/maxdB; //rescale to U8

			RF_Out[idx + currentScanline*imageLength] = sum;
	}	


		if(idx % downfactor == 0) { //Only take the even idx threads --> down sampling
			RFimage[idx / downfactor + currentScanline*imageLength] = RFimage[idx + currentScanline*imageLength];
			// 1. We take pixel idx because pixel idx is processed by this thread block, so it must be ready
			// 2. Put pixel idx = idx/2 position --> down sampling
			}

}

/* calChannelDelay
input:
------------------------------------------------------------
int imageLength		: number of pixel per scan-line
int numChannel		: number of channel of the transducer
float * tdrGeometry	: transducer geometry
int fs				: sampling frequency
int sos				: speed of sound
float time_start	: delay for RF line start
------------------------------------------------------------

output:
------------------------------------------------------------
float * channelDelay: array holding delay of each channel
------------------------------------------------------------
*/
__global__ void calChannelDelay(float * channelDelay, int imageLength, int numChannel, float * tdrGeometry, int fs, int sos, float time_start)
{
	volatile int idx = blockIdx.y * blockDim.x + threadIdx.x;
	float depth;
	float extra_depth = tdrGeometry[numChannel*2];
	if (idx<imageLength) {
		// the idx is within the image array limit
		depth = (idx*1.f*sos/fs + time_start*sos)/2;
		for (int currentChannel=0;currentChannel<numChannel;currentChannel++)
		{
			channelDelay[imageLength*currentChannel + idx] = sqrtf( tdrGeometry[currentChannel]*tdrGeometry[currentChannel] + (depth + extra_depth + tdrGeometry[numChannel + currentChannel])*(depth + extra_depth + tdrGeometry[numChannel + currentChannel]) );
		}
	}
}

/* demodulation
input:
------------------------------------------------------------
short * RFdata		: input RF data
int dataVectorLength: number of sample per received channel
int numScanline		: number of scanline acquired
int numChannel		: number of channel of the transducer
float * filter		: filter coefficient for analytic signal conversion
int FILTERLENGTH	: number of filter coefficients
------------------------------------------------------------

output:
------------------------------------------------------------
short * RFdata_I		: beamformed image
short * RFdata_Q		: beamformed image
------------------------------------------------------------

*/

__global__ void demodulation (short * RFdata_I, short * RFdata_Q, short * RFdata, int dataVectorLength, int numScanline, int numChannel, float * filter, int FILTERLENGTH)
{
	//__shared__ float coeff[FILTERLENGTH];
	extern __shared__ float coeff[];
	if (threadIdx.x < FILTERLENGTH) {
		// the thread ID is between 0 and filter order - 1
		coeff[threadIdx.x] = filter[threadIdx.x];
	}
	__syncthreads();

	volatile int z = 0;
	volatile int i = 0;
	volatile float sum_hil = 0;
	volatile int j = 0;
	volatile int pos = (blockIdx.y*gridDim.x+blockIdx.x)*dataVectorLength;

	for (i=0;i<(dataVectorLength/blockDim.x+1);i++) {
		z = i*blockDim.x + threadIdx.x;
		sum_hil=0;
		if (z<(dataVectorLength-FILTERLENGTH-1)) {
			for (j=0;j<FILTERLENGTH;j++) {
				sum_hil += coeff[j]*(float) RFdata[z + pos + j];
			}
			RFdata_I [z + pos + FILTERLENGTH/2] = (short) RFdata [z + pos + FILTERLENGTH/2];
			RFdata_Q [z + pos + FILTERLENGTH/2] = -sum_hil;
			//printf("sum_hil = %.4f\n",sum_hil);
		}
	}
}



/* Bandpass filter data using FIR coefficients
input:
------------------------------------------------------------
short * RFdata		: input RF data
int dataVectorLength: number of sample per received channel
int numScanline		: number of scanline acquired
int numChannel		: number of channel of the transducer
float * BPfilter	: filter coefficient for analytic signal conversion
int BPFILTERLENGTH	: number of coefficients in filter
------------------------------------------------------------

output:
------------------------------------------------------------
short * RFdata_Out		: filtered RF
------------------------------------------------------------

*/


__global__ void filterRF(short * RFdata_Out, short * RFdata, int dataVectorLength, int numScanline, int numChannel, float * BPfilter, int BPFILTERLENGTH)
{

	//__shared__ float coeff[BPFILTERLENGTH];
	extern __shared__ float coeff[];

	if (threadIdx.x < BPFILTERLENGTH) {
		// the thread ID is between 0 and filter order - 1
		coeff[threadIdx.x] = BPfilter[threadIdx.x];
	}
	__syncthreads();

	volatile int z = 0;
	volatile int i = 0;
	volatile float sum_BPfilter = 0;
	volatile int j = 0;
	volatile int pos = (blockIdx.y*gridDim.x + blockIdx.x)*dataVectorLength;

	for (i = 0; i < (dataVectorLength / blockDim.x + 1); i++) {
		z = i*blockDim.x + threadIdx.x;
		sum_BPfilter = 0;
		if (z < (dataVectorLength - BPFILTERLENGTH - 1)) {
			for (j = 0; j < BPFILTERLENGTH; j++) {
				sum_BPfilter += coeff[j] * (float)RFdata[z + pos + j];
			}
			RFdata_Out[z + pos + BPFILTERLENGTH / 2] = sum_BPfilter;
			//printf("sum_hil = %.4f\n",sum_hil);
		}
	}
}



int SAbeamformerFilter(short * hostRF_Out, short * hostRFimage, short * hostRFdata, int numScanline, int dataVectorLength, float * hostTdrGeometry, int numChannel, int * hostTxPos, int * hostRxPos, int numTxRxpair, float * hostWeighting, int fs, int sos, float * hostTimeStart, float * hostBPfilter, int DataFlag, float * hostfilter, int BPFILTERLENGTH, int FILTERLENGTH, int downfactor, int ADbits)
{
	// Select GPU - Default: choose cuda device 0 (you can use deviceQuery to check which card is device 0)
	hipSetDevice(0);

	// Step 1. ALLOCATE ENOUGH MEMORY (both Host and GPU)
	// 1a. Host:
	// Filter
	
	//Create Console window for debug
#ifdef DEBUG_MODE
	AllocConsole();
	
#endif

#ifdef DEBUG_MODE
	_cprintf("filterSize = %d\n", filterSize);
	_cprintf("hosfilter...%.4f\n",hostfilter[26]);
#endif

	// 1b. GPU:

	short * RFdata;
	short *RFdata_I, * RFdata_Q, *RFimage, *RFdata_Out, *RF_Out;
	float * filter, *BPfilter, * tdrGeometry, * channelDelay, * weighting, *time_start;
	int * txPos, * rxPos;
	//int dataSize_char = dataVectorLength*numScanline*numChannel*numChannel*sizeof(char);
	int dataSize_short = dataVectorLength*numScanline*numChannel*numChannel*sizeof(short);
	int imageLength = dataVectorLength;
	//_cprintf("image length = %d\n", imageLength);
	int imageSize = imageLength*numScanline*sizeof(short);
	int tdrGeometrySize = (2*numChannel+1)*sizeof(float);
	int channelDelaySize = numTxRxpair * imageLength * sizeof(float);
	int posSize = numTxRxpair * sizeof(int);

	//float ADbits = 8.0; // will eventually want to moves this as an external input.
	float maxdB = 20.f*log10f(numTxRxpair * pow(2.0, (ADbits-1)))/255; //diviser to form 8 bit data scale where 255 is max possible bit value.

	int filterSize = FILTERLENGTH * sizeof(float);
	int BPfilterSize = BPFILTERLENGTH * sizeof(float);

	// handle arbitrary number of coefficients

	//int filterSize = sizeof(hostfilter);
	//int BPfilterSize = sizeof(hostBPfilter);
	
	//int FILTERLENGTH = sizeof(hostBPfilter)/ sizeof(float);
	//int BPFILTERLENGTH = sizeof(hostBPfilter)/ sizeof(float);



	// RFdata
	hipMalloc( (void **) &RFdata, dataSize_short);
	// RFdata_I
	hipMalloc( (void **) &RFdata_I, dataSize_short);
	// RFdata_Q
	hipMalloc( (void **) &RFdata_Q, dataSize_short);
	// RFdata_Out
	hipMalloc((void **) &RFdata_Out, dataSize_short);
	// RFimage
	hipMalloc( (void **) &RFimage, imageSize);
	// RF_Out
	hipMalloc((void **) &RF_Out, imageSize);
	// filter
	hipMalloc( (void **) &filter, filterSize);
	// BPfilter
	hipMalloc((void **)&BPfilter, BPfilterSize);
	// tdrGeometry
	hipMalloc( (void **) &tdrGeometry, tdrGeometrySize );
	// channelDelay
	hipMalloc( (void **) &channelDelay, channelDelaySize);
	// weighting
	hipMalloc( (void **) &weighting, posSize); // weighting must match with the number of TxRxpair
	// txPos
	hipMalloc( (void **) &txPos, posSize);
	// rxPos
	hipMalloc( (void **) &rxPos, posSize);
	// time start
	hipMalloc( (void **) &time_start, posSize);


	hipMemset( RFdata_I, 0, dataSize_short);
	hipMemset( RFdata_Q, 0, dataSize_short);
	hipMemset( RFdata_Out, 0, dataSize_short);
	hipMemset( RFimage, 0, imageSize);
	hipMemset(RF_Out, 0, imageSize);
	hipMemset(tdrGeometry, 0, tdrGeometrySize);
	hipMemset(weighting, 0, posSize);
	hipMemset(txPos, 0, posSize);
	hipMemset(rxPos, 0, posSize);
	hipMemset(time_start, 0, posSize);




#ifdef DEBUG_MODE
	_cprintf("Step 1 done\n");
#endif

	// Step 2. COPY INPUT DATA TO GPU
	// 2a. load ASC filter from file
	hipMemcpy(filter, hostfilter, filterSize, hipMemcpyHostToDevice);
	// 2aa. load BPfilter
	hipMemcpy(BPfilter, hostBPfilter, BPfilterSize, hipMemcpyHostToDevice);
	// 2b. Copy input data to GPU
	hipMemcpy(RFdata, hostRFdata, dataSize_short, hipMemcpyHostToDevice);
	// 2c. Copy tdrGeometry
	hipMemcpy(tdrGeometry, hostTdrGeometry, tdrGeometrySize, hipMemcpyHostToDevice);
	// 2d. Copy txPos
	hipMemcpy(txPos, hostTxPos, posSize, hipMemcpyHostToDevice);
	// 2e. Copy rxPos
	hipMemcpy(rxPos, hostRxPos, posSize, hipMemcpyHostToDevice);
	// 2f. Copy weighting
	hipMemcpy(weighting, hostWeighting, posSize, hipMemcpyHostToDevice);
	// 2g. Copy time_start
	hipMemcpy(time_start, hostTimeStart, posSize, hipMemcpyHostToDevice);






#ifdef DEBUG_MODE
	_cprintf("Step 2 done\n");
#endif

	dim3 dimGridASC(numScanline, numTxRxpair);


	// Step 3b. PERFORM  FILTERING Third <<>> parameter is size of shared memory

		filterRF <<< dimGridASC, ASCBLOCKSIZE, BPFILTERLENGTH*sizeof(float)>>> (RFdata_Out, RFdata, dataVectorLength, numScanline, numChannel, BPfilter, BPFILTERLENGTH);

#ifdef DEBUG_MODE
	FILE *fp_mean;
	fp_mean = fopen("meanVector.bin","wb+");
	float * hostMeanVector;
	hostMeanVector = (float *) malloc ( numScanline * numTxRxpair * sizeof(float) );
	hipMemcpy(hostMeanVector, meanVector, numScanline * numTxRxpair * sizeof(float), hipMemcpyDeviceToHost );
	fwrite(hostMeanVector, 1, sizeof(float) * numScanline * numTxRxpair, fp_mean);
	fclose(fp_mean);
#endif

	// Step 3b. PERFORM ANALYTIC SIGNAL CONVERSION
	demodulation <<< dimGridASC, ASCBLOCKSIZE , FILTERLENGTH*sizeof(float) >>> (RFdata_I, RFdata_Q, RFdata_Out, dataVectorLength, numScanline, numChannel, filter, FILTERLENGTH);

#ifdef DEBUG_MODE
	_cprintf("Step 3 done\n");
#endif
	
	/*
	// save ASC output for debugging
	short *hostRFdata_I;
	hostRFdata_I = (short *) malloc (dataSize_short);
	hipMemcpy(hostRFdata_I, RFdata_Q, dataSize_short, hipMemcpyDeviceToHost);
	fp = fopen("RFdata_Q.bin","wb+");
	fwrite(hostRFdata_I, 1, dataSize_short, fp);
	fclose(fp);

	free(hostRFdata_I);
	*/
	
	// Step 4a. CALCULATE DELAYS FOR EACH CHANNEL
	int tmpSize =  ceilf( (float) imageLength/BFBLOCKSIZE );
	dim3 dimGridCalDelay(1, tmpSize );
	calChannelDelay <<< dimGridCalDelay, BFBLOCKSIZE >>> (channelDelay, imageLength, numChannel, tdrGeometry, fs, sos, hostTimeStart[0]);

#ifdef DEBUG_MODE
	_cprintf("Step 4a done\n");
#endif

	/*
	// save channelDelay for debugging
	float *hostDelay;
	hostDelay = (float *) malloc (channelDelaySize);
	hipMemcpy(hostDelay, channelDelay, channelDelaySize, hipMemcpyDeviceToHost);
	fp = fopen("channelDelay.bin","wb+");
	fwrite(hostDelay, 1, channelDelaySize, fp);
	fclose(fp);

	free(hostDelay);
	*/

	// Step 4b. PERFORM SA BEAMFORMING
	dim3 dimGridBF(numScanline, tmpSize );
	beamforming <<< dimGridBF, BFBLOCKSIZE >>> (RF_Out, RFimage, RFdata_I, RFdata_Q, dataVectorLength, channelDelay, numChannel, txPos, rxPos, numTxRxpair, weighting, fs, imageLength, sos, time_start, DataFlag, maxdB,downfactor);

#ifdef DEBUG_MODE
	_cprintf("Step 4b done\n");
#endif

	// Step 5. COPY IMAGE TO HOST
	hipMemcpy(hostRFimage, RFimage, imageSize, hipMemcpyDeviceToHost);
	hipMemcpy(hostRF_Out, RF_Out, imageSize, hipMemcpyDeviceToHost);

	// save RF image for debugging
#ifdef DEBUG_MODE
	fp = fopen("RFimage.bin","wb+");
	fwrite(hostRFimage, 1, imageSize, fp);
	fclose(fp);
#endif

	// Step 6. FREE UP MEMORY
	// 6a. Host memory
	//free(hostfilter);

	// 6b. GPU memory
	hipFree(RFdata);
	hipFree(RFdata_I);
	hipFree(RFdata_Q);
	hipFree(RFdata_Out);
	hipFree(RFimage);
	hipFree(RF_Out);
	hipFree(filter);
	hipFree(BPfilter);
	hipFree(tdrGeometry);
	hipFree(channelDelay);
	hipFree(txPos);
	hipFree(rxPos);
	hipFree(weighting);
	hipFree(time_start);


#ifdef DEBUG_MODE
	Sleep(3000);
	FreeConsole();
#endif

	return 0;	

}
